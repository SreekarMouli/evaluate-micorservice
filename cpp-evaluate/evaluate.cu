#include "hip/hip_runtime.h"
/* Evaluate Service written in cuda-cpp for execution on Nvidia GPUs  */
#include <iostream>
#include <nlohmann/json.hpp>
#include <hip/hip_runtime.h>

using json = nlohmann::json;

__device__ float plus(float a, float b) {
    return a + b;
}

__device__ float minus(float a, float b) {
    return a - b;
}

__device__ float multiply(float a, float b) {
    return a * b;
}

__device__ float handle_variable(const json& row, const std::string& key) {
    if (row.find(key) != row.end()) {
        return row[key].get<float>();
    }
    return 0.0; // Default value if variable is not found
}

template <typename T>
__global__ void evaluateFormula(const char* formulaJson, const char* rowsJson, T* results, int numRows) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < numRows) {
        json ast = json::parse(formulaJson);
        json row = json::parse(rowsJson); // Parse the row as a JSON object

        // Parse the JSON AST dynamically and evaluate the formula for row 'tid'
        T result = 0.0; // Initialize with the neutral element

        // Traverse the JSON AST and apply operations dynamically
        for (const auto& item : ast) {
            if (item.is_object()) {
                const std::string& operator_str = item.begin().key();
                const json& operands = item.begin().value();

                if (operator_str == "+") {
                    result = plus(result, operands[0].is_string() ? handle_variable(row, operands[0]) : operands[0].get<float>());
                } else if (operator_str == "-") {
                    result = minus(result, operands[0].is_string() ? handle_variable(row, operands[0]) : operands[0].get<float>());
                } else if (operator_str == "*") {
                    result = multiply(result, operands[0].is_string() ? handle_variable(row, operands[0]) : operands[0].get<float>());
                }
            }
        }

        results[tid] = result; // Store the result in the results array
    }
}

int main() {
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Define problem size
    int numRows = 1; // Adjust as needed
    int blockSize = 1; // Adjust as needed

    // Allocate memory on the CPU for results
    float* results = new float[numRows];

    // Convert the JSON AST and row data to strings
    std::string formulaJson = "{\"+\": [\"a\", {\"-\": [\"b\", {\"*\": [\"c\", \"d\"]}]}]}";
    std::string rowJson = "{\"a\": 1.0, \"b\": 2.0, \"c\": 3.0, \"d\": 4.0}";

    const char* formulaJsonCStr = formulaJson.c_str();
    const char* rowJsonCStr = rowJson.c_str();

    // Allocate memory on the GPU for results
    float* d_results;
    cudaStatus = hipMalloc((void**)&d_results, numRows * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memory allocation failed!" << std::endl;
        delete[] results;
        return 1;
    }

    // Launch the CUDA kernel
    evaluateFormula<float><<<numRows, blockSize>>>(formulaJsonCStr, rowJsonCStr, d_results, numRows);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_results);
        delete[] results;
        return 1;
    }

    // Copy results from GPU to CPU
    cudaStatus = hipMemcpy(results, d_results, numRows * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_results);
        delete[] results;
        return 1;
    }

    // Print or use the result
    std::cout << "Result: " << results[0] << std::endl;

    // Cleanup
    hipFree(d_results);
    delete[] results;

    return 0;
}
